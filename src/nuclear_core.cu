#include "hip/hip_runtime.h"
#include "nuclear_core.hpp"

__global__ void _cu_do_add(const int* inputA, const int* inputB, int* output) {
	*output = *inputA + *inputB;
}


int do_add(int inputA, int inputB){

	int  output;
	int *dev_inputA, *dev_inputB, *dev_Output;
	
	hipMalloc((void**)&dev_inputA, sizeof(int));
	hipMalloc((void**)&dev_inputB, sizeof(int));
	hipMalloc((void**)&dev_Output, sizeof(int));

	hipMemcpy(dev_inputA, &inputA, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_inputB, &inputB, sizeof(int), hipMemcpyHostToDevice);

	_cu_do_add<<<1, 1>>>(dev_inputA, dev_inputB, dev_Output);

	hipMemcpy(&output, dev_Output, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_inputA);
	hipFree(dev_inputB);
	hipFree(dev_Output);

	return output;
}
