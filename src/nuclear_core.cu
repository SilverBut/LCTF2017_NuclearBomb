#include "hip/hip_runtime.h"
#include "nuclear_core.hpp"

#define ror32(p,q) ( ( (p) >> (q) ) | ( (p) << ( 32 - (q) ) ) )
#define assert_cuda() do {	\
	assert(!hipGetLastError());\
	hipDeviceSynchronize();	\
} while(0);

typedef unsigned long uint32_t;
#include "flag.hpp"
#include <cassert>

__global__ void _do_bomb(const unsigned char* gpu_sbox, unsigned char* stream){
  int blockId = blockIdx.x + blockIdx.y * gridDim.x  
                   + gridDim.x * gridDim.y * blockIdx.z;  
  int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)  
                     + (threadIdx.z * (blockDim.x * blockDim.y))  
                     + (threadIdx.y * blockDim.x) + threadIdx.x;  	
	unsigned int *stream_int = (unsigned int*)stream;
	// For each int, ROR a value
	unsigned char ror_value = threadIdx.x ^ threadIdx.y;
	if ( threadId % 4 == 0 ){
		stream_int[threadId/4]=ror32(stream_int[threadId/4], ror_value);
	}
	stream[threadId] = gpu_sbox[stream[threadId]];
}

int do_bomb(unsigned int* stream){

	unsigned char* dev_stream;
	unsigned char* dev_sbox;

	hipMalloc((void**)&dev_stream, table_size * table_nlen * 4);
	hipMemcpy(dev_stream, stream, table_size*table_nlen*4, hipMemcpyHostToDevice);
	assert_cuda();

	hipMalloc((void**)&dev_sbox, 256);
	hipMemcpy(dev_sbox, sbox, 256, hipMemcpyHostToDevice);
	assert_cuda();

	_do_bomb<<< dim3(4,table_nlen,1), dim3(table_block_width, table_block_width, 1) >>>(dev_sbox, dev_stream);
	assert_cuda();

	hipMemcpy(stream, dev_stream, table_size*table_nlen*4, hipMemcpyDeviceToHost);
	assert_cuda();

	hipFree(dev_stream);
	hipFree(dev_sbox);
	assert_cuda();

	return 0;
}
